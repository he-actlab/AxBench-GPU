#include "hip/hip_runtime.h"
#include "../../../headers/activationFunction.h"

// Designed by: Amir Yazdanbakhsh
// Date: March 26th - 2015
// Alternative Computing Technologies Lab.
// Georgia Institute of Technology


#include "stdlib.h"
#include <fstream>
#include <iostream>
#include <cstddef>

// Cuda Libraries
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define MAX_LOOP 1000
#define MAX_DIFF 0.15f
#define NUM_JOINTS 3
#define PI 3.14159265358979f
#define NUM_JOINTS_P1 (NUM_JOINTS + 1)

using namespace std;

__global__ void invkin_kernel(float *xTarget_in, float *yTarget_in, float *angles, int size, float err_thresh)
{

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	if(idx < size)
	{
		float parrotInput[2];
    	float parrotOutput[3];
    	float angle_out[NUM_JOINTS];

    	for(int i = 0; i < NUM_JOINTS; i++)
    	{
  			angle_out[i] = 0.0;
    	}

    	float max_err 	= err_thresh * (float)(NUM_JOINTS);
    	float err 		= max_err + 1.f; // initialize error to something greater than error threshold

    	parrotInput[0] = xTarget_in[idx];
    	parrotInput[1] = yTarget_in[idx];

float layer_1_0 = parrotInput[0] * -1.798665 + parrotInput[1] * 4.560411 + 1.0f * 1.149290;

float layer_1_1 = parrotInput[0] * -1.262481 + parrotInput[1] * -3.736870 + 1.0f * 0.285704;

float layer_1_2 = parrotInput[0] * -1.223271 + parrotInput[1] * -3.642362 + 1.0f * 0.143369;

float layer_1_3 = parrotInput[0] * -3.280792 + parrotInput[1] * 2.001388 + 1.0f * 1.805166;

float layer_1_4 = parrotInput[0] * -2.898585 + parrotInput[1] * 1.940286 + 1.0f * 1.642777;

float layer_1_5 = parrotInput[0] * -5.762485 + parrotInput[1] * -6.614917 + 1.0f * 0.828460;

float layer_1_6 = parrotInput[0] * -7.034237 + parrotInput[1] * 0.076823 + 1.0f * 0.421022;

float layer_1_7 = parrotInput[0] * -5.059394 + parrotInput[1] * 1.127199 + 1.0f * 0.700742;

float layer_1_8 = parrotInput[0] * -1.756325 + parrotInput[1] * 4.784623 + 1.0f * 0.978976;

float layer_1_9 = parrotInput[0] * -6.691505 + parrotInput[1] * -1.578492 + 1.0f * 0.490338;

float layer_1_10 = parrotInput[0] * -2.959693 + parrotInput[1] * 0.825397 + 1.0f * 1.853845;

float layer_1_11 = parrotInput[0] * -3.711463 + parrotInput[1] * 1.052303 + 1.0f * 1.110465;

float layer_1_12 = parrotInput[0] * -1.804598 + parrotInput[1] * 3.503114 + 1.0f * 0.957473;

float layer_1_13 = parrotInput[0] * -1.629346 + parrotInput[1] * -1.502572 + 1.0f * 2.276568;

float layer_1_14 = parrotInput[0] * -6.138941 + parrotInput[1] * -3.679501 + 1.0f * 0.655275;

float layer_1_15 = parrotInput[0] * -2.587424 + parrotInput[1] * 3.923578 + 1.0f * -0.023949;

float layer_2_0 = sigmoid(layer_1_0, 0.500000) * 0.403742 + sigmoid(layer_1_1, 0.500000) * 1.047462 + sigmoid(layer_1_2, 0.500000) * 1.048148 + sigmoid(layer_1_3, 0.500000) * 0.427120 + sigmoid(layer_1_4, 0.500000) * 0.488172 + sigmoid(layer_1_5, 0.500000) * 1.157142 + sigmoid(layer_1_6, 0.500000) * 0.916205 + sigmoid(layer_1_7, 0.500000) * 0.871136 + sigmoid(layer_1_8, 0.500000) * 0.423161 + sigmoid(layer_1_9, 0.500000) * 1.136492 + sigmoid(layer_1_10, 0.500000) * 0.436868 + sigmoid(layer_1_11, 0.500000) * 0.859039 + sigmoid(layer_1_12, 0.500000) * 0.379688 + sigmoid(layer_1_13, 0.500000) * -1.204959 + sigmoid(layer_1_14, 0.500000) * 1.671170 + sigmoid(layer_1_15, 0.500000) * 1.251833 + 1.0f * 0.308960;

layer_2_0 = linear(layer_2_0, 0.5);

float layer_2_1 = sigmoid(layer_1_0, 0.500000) * 1.686440 + sigmoid(layer_1_1, 0.500000) * 3.135052 + sigmoid(layer_1_2, 0.500000) * 3.180274 + sigmoid(layer_1_3, 0.500000) * 1.642216 + sigmoid(layer_1_4, 0.500000) * 1.486881 + sigmoid(layer_1_5, 0.500000) * 2.441879 + sigmoid(layer_1_6, 0.500000) * 1.347186 + sigmoid(layer_1_7, 0.500000) * 1.998285 + sigmoid(layer_1_8, 0.500000) * 1.717950 + sigmoid(layer_1_9, 0.500000) * 2.358272 + sigmoid(layer_1_10, 0.500000) * 1.416066 + sigmoid(layer_1_11, 0.500000) * 2.602691 + sigmoid(layer_1_12, 0.500000) * 1.568467 + sigmoid(layer_1_13, 0.500000) * 2.274102 + sigmoid(layer_1_14, 0.500000) * 2.580288 + sigmoid(layer_1_15, 0.500000) * 3.413414 + 1.0f * 0.776277;

layer_2_1 = linear(layer_2_1, 0.5);

float layer_2_2 = sigmoid(layer_1_0, 0.500000) * 6.285129 + sigmoid(layer_1_1, 0.500000) * 12.153250 + sigmoid(layer_1_2, 0.500000) * 7.822494 + sigmoid(layer_1_3, 0.500000) * 6.273594 + sigmoid(layer_1_4, 0.500000) * 6.362993 + sigmoid(layer_1_5, 0.500000) * 7.813550 + sigmoid(layer_1_6, 0.500000) * 5.648221 + sigmoid(layer_1_7, 0.500000) * 4.771950 + sigmoid(layer_1_8, 0.500000) * 6.301631 + sigmoid(layer_1_9, 0.500000) * 5.755622 + sigmoid(layer_1_10, 0.500000) * 6.252254 + sigmoid(layer_1_11, 0.500000) * 6.237138 + sigmoid(layer_1_12, 0.500000) * 6.359992 + sigmoid(layer_1_13, 0.500000) * 11.693406 + sigmoid(layer_1_14, 0.500000) * 6.609825 + sigmoid(layer_1_15, 0.500000) * 6.263258 + 1.0f * 6.270045;

layer_2_2 = linear(layer_2_2, 0.5);

parrotOutput[0] = layer_2_0;

parrotOutput[1] = layer_2_1;

parrotOutput[2] = layer_2_2;

// parrotOutput[2] = layer_2_2;
// 
// 		//float max_err = err_thresh * (float)(NUM_JOINTS);
// 		//float err = max_err + 1.f;
// 
// 		// Initialize x and y data
// 		float xData[NUM_JOINTS_P1];
// 		float yData[NUM_JOINTS_P1];
// 
// 		for (int i = 0 ; i < NUM_JOINTS_P1; i++)
// 		{
// 			xData[i] = i;
// 			yData[i] = 0.f;
// 		}
// 
// 		for(int curr_loop = 0; curr_loop < MAX_LOOP; curr_loop++)
// 		{
// 			for (int iter = NUM_JOINTS; iter > 0; iter--)
// 			{
// 				float pe_x = xData[NUM_JOINTS];
// 				float pe_y = yData[NUM_JOINTS];
// 				float pc_x = xData[iter-1];
// 				float pc_y = yData[iter-1];
// 				float diff_pe_pc_x = pe_x - pc_x;
// 				float diff_pe_pc_y = pe_y - pc_y;
// 				float diff_tgt_pc_x = xTarget_in[idx] - pc_x;
// 				float diff_tgt_pc_y = yTarget_in[idx] - pc_y;
// 				float len_diff_pe_pc = sqrt(diff_pe_pc_x * diff_pe_pc_x + diff_pe_pc_y * diff_pe_pc_y);
// 				float len_diff_tgt_pc = sqrt(diff_tgt_pc_x * diff_tgt_pc_x + diff_tgt_pc_y * diff_tgt_pc_y);
// 				float a_x = diff_pe_pc_x / len_diff_pe_pc;
// 				float a_y = diff_pe_pc_y / len_diff_pe_pc;
// 				float b_x = diff_tgt_pc_x / len_diff_tgt_pc;
// 				float b_y = diff_tgt_pc_y / len_diff_tgt_pc;
// 				float a_dot_b = a_x * b_x + a_y * b_y;
// 				if (a_dot_b > 1.f)
// 					a_dot_b = 1.f;
// 				else if (a_dot_b < -1.f)
// 					a_dot_b = -1.f;
// 				float angle = acos(a_dot_b) * (180.f / PI);
// 				// Determine angle direction
// 				float direction = a_x * b_y - a_y * b_x;
// 				if (direction < 0.f)
// 					angle = -angle;
// 				// Make the result look more natural (these checks may be omitted)
// 				// if (angle > 30.f)
// 				// 	angle = 30.f;
// 				// else if (angle < -30.f)
// 				// 	angle = -30.f;
// 				// Save angle
// 				angle_out[iter - 1] = angle;
// 				for (int i = 0; i < NUM_JOINTS; i++)
// 				{
// 					if(i < NUM_JOINTS - 1)
// 					{
// 						angle_out[i+1] += angle_out[i];
// 					}
// 				}
// 			}
// 		}
// 		parrotOutput[0] = angle_out[0] / 30.0;
// 		parrotOutput[1] = angle_out[1] / 30.0;
// 		parrotOutput[2] = angle_out[2] / 30.0;
// 
// #pragma parrot(output, "invkin_kernel", [3]<-1.0; 1.0>parrotOutput)

		angle_out[0] = parrotOutput[0] * 30.0;
		angle_out[1] = parrotOutput[1] * 30.0;
		angle_out[2] = parrotOutput[2] * 30.0;

		angles[idx * NUM_JOINTS + 0] = angle_out[0];
		angles[idx * NUM_JOINTS + 1] = angle_out[1];
		angles[idx * NUM_JOINTS + 2] = angle_out[2];
	}
}
int main(int argc, char* argv[])
{
	if(argc != 4)
	{
		std::cerr << "Usage: ./invkin.out <input file coefficients> <output file> <error threshold>" << std::endl;
		exit(EXIT_FAILURE);
	}

	float* xTarget_in_h;
	float* yTarget_in_h;
	float* angle_out_h;

	hipError_t cudaStatus;

	int data_size = 0;

	// process the files
	ifstream coordinate_in_file (argv[1]);
	ofstream angle_out_file (argv[2]);
	float err_thresh = atof(argv[3]);


	if(coordinate_in_file.is_open())
	{
		coordinate_in_file >> data_size;
		std::cout << "# Data Size = " << data_size << std::endl;
	}

	// allocate the memory
	xTarget_in_h = new (nothrow) float[data_size];
	if(xTarget_in_h == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	yTarget_in_h = new (nothrow) float[data_size];
	if(yTarget_in_h == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	angle_out_h = new (nothrow) float[data_size*NUM_JOINTS];
	if(angle_out_h == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}


	// Prepare
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// add data to the arrays
	float xTarget_tmp, yTarget_tmp;
	int coeff_index = 0;
	while(coeff_index < data_size)
	{
		coordinate_in_file >> xTarget_tmp >> yTarget_tmp;

		for(int i = 0; i < NUM_JOINTS ; i++)
		{
			angle_out_h[coeff_index * NUM_JOINTS + i] = 0.0;
		}

		xTarget_in_h[coeff_index] = xTarget_tmp;
		yTarget_in_h[coeff_index++] = yTarget_tmp;
	}


	std::cout << "# Coordinates are read from file..." << std::endl;

	// memory allocations on the host
	float 	*xTarget_in_d,
			*yTarget_in_d;
	float 	*angle_out_d;

	hipMalloc((void**) &xTarget_in_d, data_size * sizeof(float));
	hipMalloc((void**) &yTarget_in_d, data_size * sizeof(float));
	hipMalloc((void**) &angle_out_d,  data_size * NUM_JOINTS * sizeof(float));

	std::cout << "# Memory allocation on GPU is done..." << std::endl;

	hipMemcpy(xTarget_in_d, xTarget_in_h, data_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yTarget_in_d, yTarget_in_h, data_size * sizeof(float), hipMemcpyHostToDevice);

	std::cout << "# Data are transfered to GPU..." << std::endl;

	dim3 dimBlock	( 512, 1 );
	dim3 dimGrid	( data_size / 512, 1 );


	hipEventRecord(start, 0);

#pragma parrot.start("invkin_kernel")

	invkin_kernel<<<dimGrid, dimBlock>>>(xTarget_in_d, yTarget_in_d, angle_out_d, data_size, err_thresh);

#pragma parrot.end("invkin_kernel")

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
       	std::cout << "Something was wrong! Error code: " << cudaStatus << std::endl;
    }

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "# Elapsed Time in `nrpoly3` kernel = " << elapsedTime << std::endl;
	std::cout << "# GPU computation is done ..." << std::endl;

	hipMemcpy(angle_out_h, angle_out_d, data_size * NUM_JOINTS * sizeof(float), hipMemcpyDeviceToHost);

	for(int i = 0; i < data_size; i++)
	{
		angle_out_file << xTarget_in_h[i] << " " << yTarget_in_h[i] << " ";
		for(int j = 0 ; j < NUM_JOINTS; j++)
		{
			angle_out_file << angle_out_h[i * NUM_JOINTS + j] << " ";
		}
		angle_out_file << std::endl;
	}

	// close files
	coordinate_in_file.close();
	angle_out_file.close();

	// de-allocate the memory
	delete[] xTarget_in_h;
	delete[] yTarget_in_h;
	delete[] angle_out_h;

	// de-allocate cuda memory
	hipFree(xTarget_in_d);
	hipFree(yTarget_in_d);
	hipFree(angle_out_d);

	std::cout << "Thank you..." << std::endl;
}
