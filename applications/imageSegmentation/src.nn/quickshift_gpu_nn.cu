#include "hip/hip_runtime.h"
#include "../../../headers/activationFunction.h"

/** @internal
 ** @file:       quickshift.cpp
 ** @author:     Brian Fulkerson
 ** @author:     Andrea Vedaldi
 ** @brief:      Quickshift command line
 **/

#include <math.h>
#include <string.h>
#include <assert.h>
#include <stdio.h>
#include "quickshift_common.h"
#include <cutil_inline.h>

texture<float, 3, hipReadModeElementType> texI;
texture<float, 2, hipReadModeElementType> texE;

#define USE_TEX_E 1
#define USE_TEX_I 1

#if USE_TEX_I
  #define TEXI(x,y,c) tex3D(texI, x + 0.5f, y + 0.5f, c + 0.5f)
#else
  #define TEXI(x,y,c) I [ (x) + N1*(y) + N2*N1*k ]
#endif

#if USE_TEX_E
  #define TEXE(x,y) tex2D(texE, x + 0.5f, y + 0.5f)
#else
  #define TEXE(x,y) E [ (x) + N1* (y)]
#endif

#define distance(I,N1,N2,K,v,j1,j2,dist)      \
{                                             \
  dist = 0 ;                                  \
  int d1 = j1 - i1 ;                          \
  int d2 = j2 - i2 ;                          \
  int k ;                                     \
  dist += d1*d1 + d2*d2 ;                     \
  for (k = 0 ; k < K ; ++k) {                 \
    float d =  v[k] - TEXI(j1,j2,k);          \
    dist += d*d ;                             \
  }                                           \
}



__device__  float myDistance(const float * I, int K, float* v, int i1, int i2, int j1, int j2)
{
  float dist = 0.0;
  for (int k = 0; k < K ; ++k)
  {
    float d = v[k] - TEXI(j1, j2, k);
    dist += d*d;
  }
  return dist;
}


extern "C"
int iDivUp(int num, int denom)
{
  return (num % denom != 0) ? (num / denom + 1) : (num / denom);
}


extern "C"
__global__ void find_neighbors_gpu(const float * I, int N1, int N2, int K, float * E, float tau2, int tR, float * map, float * gaps)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (i1 >= N1 || i2 >= N2) return; // out of bounds

  int j1,j2;

  /* Quickshift assigns each i to the closest j which has an increase in the
   * density (E). If there is no j s.t. Ej > Ei, then gaps_i == inf (a root
   * node in one of the trees of merges).
   */

  float E0 = TEXE(i1, i2) ;
  float d_best = INF ;
  float j1_best = i1   ;
  float j2_best = i2   ;

  int j1min = VL_MAX(i1 - tR, 0   ) ;
  int j1max = VL_MIN(i1 + tR, N1-1) ;
  int j2min = VL_MAX(i2 - tR, 0   ) ;
  int j2max = VL_MIN(i2 + tR, N2-1) ;

  /* Cache the center value in registers */
  float v[3];
  for (int k = 0 ; k < K ; ++k) {
    v[k] =  TEXI(i1,i2,k);
    }

  for (j2 = j2min ; j2 <= j2max ; ++ j2) {
    for (j1 = j1min ; j1 <= j1max ; ++ j1) {
      if (TEXE(j1,j2) > E0) {
        float Dij;
        distance(I,N1,N2,K, v, j1,j2,Dij) ;
        if (Dij <= tau2 && Dij < d_best) {
          d_best = Dij ;
          j1_best = j1 ;
          j2_best = j2 ;
        }
      }
    }
  }

  /* map is the index of the best pair */
  /* gaps_i is the minimal distance, inf implies no Ej > Ei within
   * distance tau from the point */
  map [i1 + N1 * i2] = j1_best + N1 * j2_best ; /* + 1 ; */
  if (map[i1 + N1 * i2] != i1 + N1 * i2)
    gaps[i1 + N1 * i2] = sqrt(d_best) ;
  else
    gaps[i1 + N1 * i2] = d_best; /* inf */
}

extern "C"
__global__ void compute_E_gpu(const float * I, int N1, int N2, int K, int R, float
    sigma, float * E, float * n, float * M)
{
  int i1 = blockIdx.y * blockDim.y + threadIdx.y;
  int i2 = blockIdx.x * blockDim.x + threadIdx.x;
  if (i1 >= N1 || i2 >= N2) return; // out of bounds
  int j1,j2;

  /* -----------------------------------------------------------------
   *                                                 E = - [oN'*F]', M
   * -------------------------------------------------------------- */

  /*
     D_ij = d(x_i,x_j)
     E_ij = exp(- .5 * D_ij / sigma^2) ;
     F_ij = - E_ij
     E_i  = sum_j E_ij
     M_di = sum_j X_j F_ij

     E is the parzen window estimate of the density
     0 = dissimilar to everything, windowsize = identical
  */

  int j1min = VL_MAX(i1 - R, 0   ) ;
  int j1max = VL_MIN(i1 + R, N1-1) ;
  int j2min = VL_MAX(i2 - R, 0   ) ;
  int j2max = VL_MIN(i2 + R, N2-1) ;
  float Ei = 0;

  /* Cache the center value in registers */
  float v[3];
  for (int k = 0 ; k < K ; ++k) {
    v[k] =  TEXI(i1,i2,k);
    }


  // Amir
  float parrotInput[6];
  float parrotOutput[1];
  // Rima

  /* For each pixel in the window compute the distance between it and the
   * source pixel */
  for (j2 = j2min ; j2 <= j2max ; ++ j2) {
    for (j1 = j1min ; j1 <= j1max ; ++ j1) {
      float Dij;


      //distance(I, N1, N2, K,v ,j1, j2, Dij) ;
      parrotInput[0] = v[0];
      parrotInput[1] = v[1];
      parrotInput[2] = v[2];
      parrotInput[3] = TEXI(j1,j2,0);
      parrotInput[4] = TEXI(j1,j2,1);
      parrotInput[5] = TEXI(j1,j2,2);
      float Fij;

float layer_1_0 = parrotInput[0] * -1.522834 + parrotInput[1] * -0.662261 + parrotInput[2] * -1500.000000 + parrotInput[3] * 1500.000000 + parrotInput[4] * 1500.000000 + parrotInput[5] * 209.477997 + 1.0f * -0.517169;

float layer_1_1 = parrotInput[0] * -0.387868 + parrotInput[1] * -0.369250 + parrotInput[2] * 0.427768 + parrotInput[3] * 0.192909 + parrotInput[4] * -0.445800 + parrotInput[5] * 0.655006 + 1.0f * -1.834540;

float layer_1_2 = parrotInput[0] * 136.454300 + parrotInput[1] * -2.051326 + parrotInput[2] * 1500.000000 + parrotInput[3] * -135.596634 + parrotInput[4] * -1442.484253 + parrotInput[5] * -122.538925 + 1.0f * -1.537653;

float layer_1_3 = parrotInput[0] * -0.296651 + parrotInput[1] * 0.333198 + parrotInput[2] * -0.753976 + parrotInput[3] * -0.367202 + parrotInput[4] * -0.499004 + parrotInput[5] * 1.655033 + 1.0f * -2.038385;

float layer_2_0 = sigmoid(layer_1_0, 0.500000) * 0.001715 + sigmoid(layer_1_1, 0.500000) * 0.146615 + sigmoid(layer_1_2, 0.500000) * -0.007966 + sigmoid(layer_1_3, 0.500000) * -0.142282 + 1.0f * 0.011396;

layer_2_0 = linear(layer_2_0, 0.5);

parrotOutput[0] = layer_2_0;

// parrotOutput[0] = layer_2_0;
// 
//       Dij = myDistance(I, K, v, i1, i2, j1, j2);
//       int d1 = j1 - i1;
//       int d2 = j2 - i2;
//       Dij += d1 * d1 + d2 * d2;
//       //distance(I,N1,N2,K, v, j1,j2,Dij) ;
// 
//       /* Make distance a similarity */
//       Fij = - exp(- Dij / (2*sigma*sigma)) ;
// 
//       parrotOutput[0] =  - Fij;
// 
// #pragma parrot(output, "compute_E_gpu", [1]<-1.0; 1.0>parrotOutput)

      Fij =  - parrotOutput[0];

      /* E is E_i above */
      Ei += -Fij;

    } /* j1 */
  } /* j2 */
  /* Normalize */
  E [i1 + N1 * i2] = Ei / ((j1max-j1min)*(j2max-j2min));
}


extern "C"
void quickshift_gpu(image_t im, float sigma, float tau, float * map, float * gaps, float * E)
{
#if USE_TEX_I
  //printf("quickshiftGPU: using texture for I\n");
  hipArray * cu_array_I;

  // Allocate array
  hipChannelFormatDesc descriptionI = hipCreateChannelDesc<float>();

  hipExtent const ext = {im.N1, im.N2, im.K};
  hipMalloc3DArray(&cu_array_I, &descriptionI, ext);

  hipMemcpy3DParms copyParams = {0};
  copyParams.extent = make_hipExtent(im.N1, im.N2, im.K);
  copyParams.kind = hipMemcpyHostToDevice;
  copyParams.dstArray = cu_array_I;
  // The pitched pointer is really tricky to get right. We give the
  // pitch of a row, then the number of elements in a row, then the
  // height, and we omit the 3rd dimension.
  copyParams.srcPtr = make_hipPitchedPtr(
  (void*)&im.I[0], ext.width*sizeof(float), ext.width, ext.height);
  CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));

  CUDA_SAFE_CALL(hipBindTextureToArray(texI, cu_array_I,
        descriptionI));

  texI.normalized = false;
  texI.filterMode = hipFilterModePoint;
#endif


  float *map_d, *E_d, *gaps_d, *I;

  int verb = 1 ;

  float tau2;

  int K;
  int N1,N2, R, tR;

  N1 = im.N1;
  N2 = im.N2;
  K = im.K;

  //d = 2 + K ; /* Total dimensions include spatial component (x,y) */

  tau2  = tau*tau;

  unsigned int size = im.N1*im.N2 * sizeof(float);
  cutilSafeCall( hipMalloc( (void**) &I, size*im.K));
  cutilSafeCall( hipMalloc( (void**) &map_d, size));
  cutilSafeCall( hipMalloc( (void**) &gaps_d, size));
  cutilSafeCall( hipMalloc( (void**) &E_d, size));

  cutilSafeCall( hipMemcpy( I, im.I, size*im.K, hipMemcpyHostToDevice));
  cutilSafeCall( hipMemset( E_d, 0, size));

  R = (int) ceil (3 * sigma) ;
  tR = (int) ceil (tau) ;

  if (verb) {
    //printf("quickshiftGPU: [N1,N2,K]: [%d,%d,%d]\n", N1,N2,K) ;
    //printf("quickshiftGPU: type: quick\n");
    //printf("quickshiftGPU: sigma:   %g\n", sigma) ;
    /* R is ceil(3 * sigma) and determines the window size to accumulate
     * similarity */
    //printf("quickshiftGPU: R:       %d\n", R) ;
    //printf("quickshiftGPU: tau:     %g\n", tau) ;
    //printf("quickshiftGPU: tR:      %d\n", tR) ;
  }

  unsigned int Etimer;
  cutilCheckError( cutCreateTimer(&Etimer) );
  cutilCheckError( cutResetTimer(Etimer) );
  cutilCheckError( cutStartTimer(Etimer) );

  dim3 dimBlock(32,4,1);
  dim3 dimGrid(iDivUp(N2, dimBlock.x), iDivUp(N1, dimBlock.y), 1);

#pragma parrot.start("compute_E_gpu")

  compute_E_gpu <<<dimGrid,dimBlock>>> (I, N1, N2, K, R, sigma, E_d, 0, 0);

#pragma parrot.end("compute_E_gpu")

  cutilSafeCall( hipDeviceSynchronize() );
  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");

  cutilSafeCall( hipMemcpy(E, E_d, size, hipMemcpyDeviceToHost));

  cutilCheckError( cutStopTimer(Etimer) );
  float ETime = cutGetTimerValue(Etimer);
  //printf("ComputeE: %fms\n", ETime);

  unsigned int Ntimer;
  cutilCheckError( cutCreateTimer(&Ntimer) );
  cutilCheckError( cutResetTimer(Ntimer) );
  cutilCheckError( cutStartTimer(Ntimer) );

  /* Texture map E */
#if USE_TEX_E
  //printf("quickshiftGPU: using texture for E\n");
  hipChannelFormatDesc descriptionE = hipCreateChannelDesc<float>();

  hipArray * cu_array_E;
  hipMallocArray(&cu_array_E, &descriptionE, im.N1, im.N2);

  CUDA_SAFE_CALL(hipMemcpyToArray(cu_array_E, 0, 0, E,
        sizeof(float)*im.N1*im.N2, hipMemcpyHostToDevice));

  texE.normalized = false;
  texE.filterMode = hipFilterModePoint;

  CUDA_SAFE_CALL( hipBindTextureToArray(texE, cu_array_E,
        descriptionE));
  cutilCheckMsg("Texture setup failed");

  cutilSafeCall( hipDeviceSynchronize() );
#endif

  /* -----------------------------------------------------------------
   *                                               Find best neighbors
   * -------------------------------------------------------------- */

  find_neighbors_gpu <<<dimGrid,dimBlock>>> (I, N1 ,N2, K, E_d, tau2,
      tR, map_d, gaps_d);

  cutilSafeCall( hipDeviceSynchronize() );
  // check if kernel execution generated an error
  cutilCheckMsg("Kernel execution failed");

  hipMemcpy(map, map_d, size, hipMemcpyDeviceToHost);
  hipMemcpy(gaps, gaps_d, size, hipMemcpyDeviceToHost);

  cutilCheckError( cutStopTimer(Ntimer) );
  float NTime = cutGetTimerValue(Ntimer);
  //printf("ComputeN: %fms\n", NTime);
  //printf("dimGrid: %d %d\n", dimGrid.x, dimGrid.y);
  //printf("dimBlock: %d %d\n", dimBlock.x, dimBlock.y);

  cutilSafeCall(hipFree(I));
  cutilSafeCall(hipFree(map_d));
  cutilSafeCall(hipFree(gaps_d));
  cutilSafeCall(hipFree(E_d));

}
