#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */


#include <helper_functions.h>   // helper functions for string parsing
#include <hip/hip_runtime_api.h>        // helper functions CUDA error checking and initialization
 // Amir
#include <fstream>
 using namespace std;
// Rima
////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(
    float *h_CallResult,
    float *h_PutResult,
    float *h_StockPrice,
    float *h_OptionStrike,
    float *h_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



///////////////////////////////////////////////////////////////////////////////
// Polynomial approximation of cumulative normal distribution function
///////////////////////////////////////////////////////////////////////////////
__device__ inline float cndGPU(float d)
{
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float
    K = 1.0f / (1.0f + 0.2316419f * fabsf(d));

    float
    cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
__device__ inline void BlackScholesBodyGPU(
    float &CallResult,
    float &PutResult,
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
)
{
    float sqrtT, expRT;
    float d1, d2, CNDD1, CNDD2;

    float parrotInput[3];
    float parrotOutput[1];

    parrotInput[0] = S;
    parrotInput[1] = X;
    parrotInput[2] = T;

#pragma parrot(input, "BlackScholesBodyGPU", [3]parrotInput)
    
    sqrtT = sqrtf(T);
    d1 = (__logf(S / X) + (R + 0.5f * V * V) * T) / (V * sqrtT);
    d2 = d1 - V * sqrtT;

    CNDD1 = cndGPU(d1);
    CNDD2 = cndGPU(d2);

    //Calculate Call and Put simultaneously
    expRT = __expf(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    parrotOutput[0] = CallResult;

#pragma parrot(output, "BlackScholesBodyGPU", [1]<0.0; 1.0>parrotOutput)

    CallResult = parrotOutput[0];
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__global__ void BlackScholesGPU(
    float *d_CallResult,
    float *d_PutResult,
    float *d_StockPrice,
    float *d_OptionStrike,
    float *d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
)
{
    ////Thread index
    //const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    ////Total number of threads in execution grid
    //const int THREAD_N = blockDim.x * gridDim.x;

    const int opt = blockDim.x * blockIdx.x + threadIdx.x;

    //No matter how small is execution grid or how large OptN is,
    //exactly OptN indices will be processed with perfect memory coalescing
    //for (int opt = tid; opt < optN; opt += THREAD_N)
    if (opt < optN)
        BlackScholesBodyGPU(
            d_CallResult[opt],
            d_PutResult[opt],
            d_StockPrice[opt],
            d_OptionStrike[opt],
            d_OptionYears[opt],
            Riskfree,
            Volatility
        );
}

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;
const int  NUM_ITERATIONS = 512;


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

#pragma parrot.start("BlackScholesBodyGPU")
    // Start logs
    //printf("[%s] - Starting...\n", argv[0]);

    //'h_' prefix - CPU (host) memory space
    float
    //Results calculated by CPU for reference
    *h_CallResultCPU,
    *h_PutResultCPU,
    //CPU copy of GPU results
    *h_CallResultGPU,
    *h_PutResultGPU,
    //CPU instance of input data
    *h_StockPrice,
    *h_OptionStrike,
    *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
    //Results calculated by GPU
    *d_CallResult,
    *d_PutResult,
    //GPU instance of input data
    *d_StockPrice,
    *d_OptionStrike,
    *d_OptionYears;

    //double
    //delta, ref, sum_delta, sum_ref, max_delta, L1norm, gpuTime;
    //double gpuTime;

    //StopWatchInterface *hTimer = NULL;
    int i;

    //findCudaDevice(argc, (const char **)argv);

    //sdkCreateTimer(&hTimer);

    //printf("Initializing data...\n");
    //printf("...allocating CPU memory for options.\n");
    h_CallResultCPU = (float *)malloc(OPT_SZ);
    h_PutResultCPU  = (float *)malloc(OPT_SZ);
    h_CallResultGPU = (float *)malloc(OPT_SZ);
    h_PutResultGPU  = (float *)malloc(OPT_SZ);
    h_StockPrice    = (float *)malloc(OPT_SZ);
    h_OptionStrike  = (float *)malloc(OPT_SZ);
    h_OptionYears   = (float *)malloc(OPT_SZ);

    //printf("...allocating GPU memory for options.\n");
    checkCudaErrors(hipMalloc((void **)&d_CallResult,   OPT_SZ));
    checkCudaErrors(hipMalloc((void **)&d_PutResult,    OPT_SZ));
    checkCudaErrors(hipMalloc((void **)&d_StockPrice,   OPT_SZ));
    checkCudaErrors(hipMalloc((void **)&d_OptionStrike, OPT_SZ));
    checkCudaErrors(hipMalloc((void **)&d_OptionYears,  OPT_SZ));

    //printf("...generating input data in CPU mem.\n");
    srand(5347);

    // Amir
    std::ifstream dataFile(argv[1]);
    int numberOptions;
    dataFile >> numberOptions;
    std::cout << "Total number of options:  " << numberOptions << std::endl;
    float stockPrice, optionStrike, optionYear;
    // Rima

    //Generate options set
    for (i = 0; i < numberOptions; i++)
    {
        h_CallResultCPU[i] = 0.0f;
        h_PutResultCPU[i]  = -1.0f;
        //h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
        //h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
        //h_OptionYears[i]   = RandFloat(0.25f, 10.0f);

        // Amir
        dataFile >> stockPrice >> optionStrike >> optionYear;
        h_StockPrice[i] = stockPrice;
        h_OptionStrike[i] = optionStrike;
        h_OptionYears[i] =  optionYear;      
        // Rima
    }

    int optionSize = numberOptions * sizeof(float);

    //printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing
    checkCudaErrors(hipMemcpy(d_StockPrice,  h_StockPrice,   optionSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionStrike, h_OptionStrike,  optionSize, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_OptionYears,  h_OptionYears,   optionSize, hipMemcpyHostToDevice));
    //printf("Data init done.\n\n");



    //printf("Executing Black-Scholes GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
    checkCudaErrors(hipDeviceSynchronize());
    //sdkResetTimer(&hTimer);
    //sdkStartTimer(&hTimer);

    for (i = 0; i < NUM_ITERATIONS; i++)
    {
        BlackScholesGPU<<<DIV_UP(numberOptions, 128), 128/*480, 128*/>>>(
            d_CallResult,
            d_PutResult,
            d_StockPrice,
            d_OptionStrike,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            numberOptions
        );
        getLastCudaError("BlackScholesGPU() execution failed\n");
    }

    checkCudaErrors(hipDeviceSynchronize());
    //sdkStopTimer(&hTimer);
    //gpuTime = sdkGetTimerValue(&hTimer) / NUM_ITERATIONS;

    //Both call and put is calculated
    //printf("Options count             : %i     \n", 2 * numberOptions);
    //printf("BlackScholesGPU() time    : %f msec\n", gpuTime);
    //printf("Effective memory bandwidth: %f GB/s\n", ((double)(5 * numberOptions * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
    //printf("Gigaoptions per second    : %f     \n\n", ((double)(2 * numberOptions) * 1E-9) / (gpuTime * 1E-3));

    //printf("BlackScholes, Throughput = %.4f GOptions/s, Time = %.5f s, Size = %u options, NumDevsUsed = %u, Workgroup = %u\n",
           //(((double)(2.0 * numberOptions) * 1.0E-9) / (gpuTime * 1.0E-3)), gpuTime*1e-3, (2 * numberOptions), 1, 128);

    //printf("\nReading back GPU results...\n");
    //Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_CallResultGPU, d_CallResult, optionSize, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_PutResultGPU,  d_PutResult,  optionSize, hipMemcpyDeviceToHost));


    // printf("Checking the results...\n");
    // printf("...running CPU calculations.\n\n");
    // //Calculate options values on CPU
    // BlackScholesCPU(
    //     h_CallResultCPU,
    //     h_PutResultCPU,
    //     h_StockPrice,
    //     h_OptionStrike,
    //     h_OptionYears,
    //     RISKFREE,
    //     VOLATILITY,
    //     numberOptions
    // );

    //printf("Comparing the results...\n");
    //Calculate max absolute difference and L1 distance
    //between CPU and GPU results
    //sum_delta = 0;
    //sum_ref   = 0;
    //max_delta = 0;


    // Amir
    ofstream callResultFile;
    callResultFile.open(argv[2]);
    for (i = 0 ; i < numberOptions; i++)
    {
        callResultFile << h_CallResultGPU[i] << std::endl;
    }
    callResultFile.close();
    // Rima


    // for (i = 0; i < numberOptions; i++)
    // {
    //     ref   = h_CallResultCPU[i];
    //     delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);

    //     if (delta > max_delta)
    //     {
    //         max_delta = delta;
    //     }

    //     sum_delta += delta;
    //     sum_ref   += fabs(ref);
    // }

    //L1norm = sum_delta / sum_ref;
    //printf("L1 norm: %E\n", L1norm);
    //printf("Max absolute error: %E\n\n", max_delta);

#pragma parrot.end("BlackScholesBodyGPU")

    // printf("Shutting down...\n");
    // printf("...releasing GPU memory.\n");
    checkCudaErrors(hipFree(d_OptionYears));
    checkCudaErrors(hipFree(d_OptionStrike));
    checkCudaErrors(hipFree(d_StockPrice));
    checkCudaErrors(hipFree(d_PutResult));
    checkCudaErrors(hipFree(d_CallResult));

    //printf("...releasing CPU memory.\n");
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_PutResultGPU);
    free(h_CallResultGPU);
    free(h_PutResultCPU);
    free(h_CallResultCPU);
    //sdkDeleteTimer(&hTimer);
    //printf("Shutdown done.\n");

    //printf("\n[BlackScholes] - Test Summary\n");

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    // if (L1norm > 1e-6)
    // {
    //     printf("Test failed!\n");
    //     exit(EXIT_FAILURE);
    // }

    //printf("Test passed\n");
    exit(EXIT_SUCCESS);
}
