#include "hip/hip_runtime.h"
#include "../../../headers/activationFunction.h"

/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <helper_string.h>

#include "sobel-kernels.h"

// Texture reference for reading image
texture<unsigned char, 2> tex;
extern __shared__ unsigned char LocalBlock[];
static hipArray *array = NULL;

#define RADIUS 1

#ifdef FIXED_BLOCKWIDTH
#define BlockWidth 80
#define SharedPitch 384
#endif

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


__device__ unsigned char
ComputeSobel(unsigned char ul, // upper left
             unsigned char um, // upper middle
             unsigned char ur, // upper right
             unsigned char ml, // middle left
             unsigned char mm, // middle (unused)
             unsigned char mr, // middle right
             unsigned char ll, // lower left
             unsigned char lm, // lower middle
             unsigned char lr, // lower right
             float fScale)
{

    short Sum = 0;

    // amir - observe inputs/outputs
    float parrotInput[9];
    float parrotOutput[1];

    parrotInput[0] = ul/255.0;
    parrotInput[1] = um/255.0;
    parrotInput[2] = ur/255.0;

    parrotInput[3] = ml/255.0;
    parrotInput[4] = mm/255.0;
    parrotInput[5] = mr/255.0;

    parrotInput[6] = ll/255.0;
    parrotInput[7] = lm/255.0;
    parrotInput[8] = lr/255.0;
    // rima

float layer_1_0 = parrotInput[0] * 2.906975 + parrotInput[1] * 18.102823 + parrotInput[2] * 7.359772 + parrotInput[3] * -1.966690 + parrotInput[4] * -0.452597 + parrotInput[5] * 1.842741 + parrotInput[6] * -7.301652 + parrotInput[7] * -18.254036 + parrotInput[8] * -1.580491 + 1.0f * -2.947222;

float layer_1_1 = parrotInput[0] * 10.651690 + parrotInput[1] * -2.859231 + parrotInput[2] * -1.752389 + parrotInput[3] * 8.463883 + parrotInput[4] * 2.808856 + parrotInput[5] * -10.682747 + parrotInput[6] * 1.568864 + parrotInput[7] * -0.049021 + parrotInput[8] * -7.930051 + 1.0f * -1.999825;

float layer_1_2 = parrotInput[0] * -4.211637 + parrotInput[1] * -5.409542 + parrotInput[2] * -0.236575 + parrotInput[3] * -2.193843 + parrotInput[4] * 0.163089 + parrotInput[5] * 2.582590 + parrotInput[6] * 0.226871 + parrotInput[7] * 2.421903 + parrotInput[8] * 2.988783 + 1.0f * -0.116642;

float layer_1_3 = parrotInput[0] * 12.918195 + parrotInput[1] * 0.540606 + parrotInput[2] * 0.379654 + parrotInput[3] * 0.358617 + parrotInput[4] * 0.137156 + parrotInput[5] * -8.629834 + parrotInput[6] * -0.043213 + parrotInput[7] * -2.174679 + parrotInput[8] * -2.282538 + 1.0f * -0.142923;

float layer_1_4 = parrotInput[0] * 12.283479 + parrotInput[1] * 4.163111 + parrotInput[2] * 3.605873 + parrotInput[3] * -0.038830 + parrotInput[4] * 0.467714 + parrotInput[5] * -2.938643 + parrotInput[6] * 0.100889 + parrotInput[7] * -10.761487 + parrotInput[8] * -3.418102 + 1.0f * 4.506928;

float layer_1_5 = parrotInput[0] * 11.038653 + parrotInput[1] * 0.884143 + parrotInput[2] * 0.362370 + parrotInput[3] * 0.290871 + parrotInput[4] * 0.087071 + parrotInput[5] * -7.145895 + parrotInput[6] * 0.145451 + parrotInput[7] * -3.419954 + parrotInput[8] * -1.177712 + 1.0f * 0.106418;

float layer_1_6 = parrotInput[0] * 12.201486 + parrotInput[1] * -6.125398 + parrotInput[2] * -8.649518 + parrotInput[3] * 4.240808 + parrotInput[4] * 0.245731 + parrotInput[5] * -13.141877 + parrotInput[6] * 11.406070 + parrotInput[7] * 0.852653 + parrotInput[8] * -0.399534 + 1.0f * -2.261885;

float layer_1_7 = parrotInput[0] * -6.720450 + parrotInput[1] * -6.182827 + parrotInput[2] * -0.158260 + parrotInput[3] * -6.175715 + parrotInput[4] * -0.253521 + parrotInput[5] * 4.252450 + parrotInput[6] * -0.042972 + parrotInput[7] * 8.334409 + parrotInput[8] * 7.170712 + 1.0f * -1.639585;

float layer_2_0 = sigmoid(layer_1_0, 0.500000) * 6.607004 + sigmoid(layer_1_1, 0.500000) * 4.956233 + sigmoid(layer_1_2, 0.500000) * 0.169556 + sigmoid(layer_1_3, 0.500000) * -1.821482 + sigmoid(layer_1_4, 0.500000) * -1.815212 + sigmoid(layer_1_5, 0.500000) * -1.499189 + sigmoid(layer_1_6, 0.500000) * 2.808338 + sigmoid(layer_1_7, 0.500000) * 5.343754 + 1.0f * -0.523679;

layer_2_0 = sigmoid(layer_2_0, 0.5);

parrotOutput[0] = layer_2_0;

// parrotOutput[0] = layer_2_0;
// 
//     short Horz = ur + 2*mr + lr - ul - 2*ml - ll;
//     short Vert = ul + 2*um + ur - ll - 2*lm - lr;
//     Sum = (short)(fScale*(abs((int)Horz)+abs((int)Vert)));
// 
//     if (Sum < 0)
//     {
//         Sum = 0;
//     }
//     else if (Sum > 0xff)
//     {
//         Sum = 0xff;
//     }
//     parrotOutput[0] = Sum/255.0;
// 
// #pragma parrot(output, "sobel", [1]<0.0; 1.0>parrotOutput)

    return (unsigned char) parrotOutput[0] * 255.0;
}

__global__ void
SobelShared(uchar4 *pSobelOriginal, unsigned short SobelPitch,
#ifndef FIXED_BLOCKWIDTH
            short BlockWidth, short SharedPitch,
#endif
            short w, short h, float fScale)
{
    short u = 4*blockIdx.x*BlockWidth;
    short v = blockIdx.y*blockDim.y + threadIdx.y;
    short ib;

    int SharedIdx = threadIdx.y * SharedPitch;

    for (ib = threadIdx.x; ib < BlockWidth+2*RADIUS; ib += blockDim.x)
    {
        LocalBlock[SharedIdx+4*ib+0] = tex2D(tex,
                                             (float)(u+4*ib-RADIUS+0), (float)(v-RADIUS));
        LocalBlock[SharedIdx+4*ib+1] = tex2D(tex,
                                             (float)(u+4*ib-RADIUS+1), (float)(v-RADIUS));
        LocalBlock[SharedIdx+4*ib+2] = tex2D(tex,
                                             (float)(u+4*ib-RADIUS+2), (float)(v-RADIUS));
        LocalBlock[SharedIdx+4*ib+3] = tex2D(tex,
                                             (float)(u+4*ib-RADIUS+3), (float)(v-RADIUS));
    }

    if (threadIdx.y < RADIUS*2)
    {
        //
        // copy trailing RADIUS*2 rows of pixels into shared
        //
        SharedIdx = (blockDim.y+threadIdx.y) * SharedPitch;

        for (ib = threadIdx.x; ib < BlockWidth+2*RADIUS; ib += blockDim.x)
        {
            LocalBlock[SharedIdx+4*ib+0] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+0), (float)(v+blockDim.y-RADIUS));
            LocalBlock[SharedIdx+4*ib+1] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+1), (float)(v+blockDim.y-RADIUS));
            LocalBlock[SharedIdx+4*ib+2] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+2), (float)(v+blockDim.y-RADIUS));
            LocalBlock[SharedIdx+4*ib+3] = tex2D(tex,
                                                 (float)(u+4*ib-RADIUS+3), (float)(v+blockDim.y-RADIUS));
        }
    }

    __syncthreads();

    u >>= 2;    // index as uchar4 from here
    uchar4 *pSobel = (uchar4 *)(((char *) pSobelOriginal)+v*SobelPitch);
    SharedIdx = threadIdx.y * SharedPitch;

    for (ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x)
    {

        unsigned char pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+0];
        unsigned char pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+1];
        unsigned char pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+2];
        unsigned char pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+0];
        unsigned char pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+1];
        unsigned char pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+2];
        unsigned char pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+0];
        unsigned char pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+1];
        unsigned char pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+2];

        uchar4 out;

        out.x = ComputeSobel(pix00, pix01, pix02,
                             pix10, pix11, pix12,
                             pix20, pix21, pix22, fScale);

        pix00 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+3];
        pix10 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+3];
        pix20 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+3];
        out.y = ComputeSobel(pix01, pix02, pix00,
                             pix11, pix12, pix10,
                             pix21, pix22, pix20, fScale);

        pix01 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+4];
        pix11 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+4];
        pix21 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+4];
        out.z = ComputeSobel(pix02, pix00, pix01,
                             pix12, pix10, pix11,
                             pix22, pix20, pix21, fScale);

        pix02 = LocalBlock[SharedIdx+4*ib+0*SharedPitch+5];
        pix12 = LocalBlock[SharedIdx+4*ib+1*SharedPitch+5];
        pix22 = LocalBlock[SharedIdx+4*ib+2*SharedPitch+5];
        out.w = ComputeSobel(pix00, pix01, pix02,
                             pix10, pix11, pix12,
                             pix20, pix21, pix22, fScale);

        if (u+ib < w/4 && v < h)
        {
            pSobel[u+ib] = out;
        }
    }

    __syncthreads();
}

__global__ void
SobelCopyImage(Pixel *pSobelOriginal, unsigned int Pitch,
               int w, int h, float fscale)
{
    unsigned char *pSobel =
        (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

    for (int i = threadIdx.x; i < w; i += blockDim.x)
    {
        pSobel[i] = min(max((tex2D(tex, (float) i, (float) blockIdx.x) * fscale), 0.f), 255.f);
    }
}

__global__ void
SobelTex(Pixel *pSobelOriginal, unsigned int Pitch,
         int w, int h, float fScale)
{
    unsigned char *pSobel =
        (unsigned char *)(((char *) pSobelOriginal)+blockIdx.x*Pitch);

    for (int i = threadIdx.x; i < w; i += blockDim.x)
    {
        unsigned char pix00 = tex2D(tex, (float) i-1, (float) blockIdx.x-1);
        unsigned char pix01 = tex2D(tex, (float) i+0, (float) blockIdx.x-1);
        unsigned char pix02 = tex2D(tex, (float) i+1, (float) blockIdx.x-1);
        unsigned char pix10 = tex2D(tex, (float) i-1, (float) blockIdx.x+0);
        unsigned char pix11 = tex2D(tex, (float) i+0, (float) blockIdx.x+0);
        unsigned char pix12 = tex2D(tex, (float) i+1, (float) blockIdx.x+0);
        unsigned char pix20 = tex2D(tex, (float) i-1, (float) blockIdx.x+1);
        unsigned char pix21 = tex2D(tex, (float) i+0, (float) blockIdx.x+1);
        unsigned char pix22 = tex2D(tex, (float) i+1, (float) blockIdx.x+1);
        pSobel[i] = ComputeSobel(pix00, pix01, pix02,
                                 pix10, pix11, pix12,
                                 pix20, pix21, pix22, fScale);
    }
}

extern "C" void setupTexture(int iw, int ih, Pixel *data, int Bpp)
{
    hipChannelFormatDesc desc;

    if (Bpp == 1)
    {
        desc = hipCreateChannelDesc<unsigned char>();
    }
    else
    {
        desc = hipCreateChannelDesc<uchar4>();
    }

    checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
    checkCudaErrors(hipMemcpyToArray(array, 0, 0, data, Bpp*sizeof(Pixel)*iw*ih, hipMemcpyHostToDevice));
}

extern "C" void deleteTexture(void)
{
    checkCudaErrors(hipFreeArray(array));
}


// Wrapper for the __global__ call that sets up the texture and threads
extern "C" void sobelFilter(Pixel *odata, int iw, int ih, enum SobelDisplayMode mode, float fScale)
{

// amir - start approximable region
#pragma parrot.start("sobel")

    checkCudaErrors(hipBindTextureToArray(tex, array));

    switch (mode)
    {
        case SOBELDISPLAY_IMAGE:
            SobelCopyImage<<<ih, 384>>>(odata, iw, iw, ih, fScale);
            break;

        case SOBELDISPLAY_SOBELTEX:
            SobelTex<<<ih, 384>>>(odata, iw, iw, ih, fScale);
            break;

        case SOBELDISPLAY_SOBELSHARED:
            {
                dim3 threads(16,4);
#ifndef FIXED_BLOCKWIDTH
                int BlockWidth = 80; // must be divisible by 16 for coalescing
#endif
                dim3 blocks = dim3(iw/(4*BlockWidth)+(0!=iw%(4*BlockWidth)),
                                   ih/threads.y+(0!=ih%threads.y));
                int SharedPitch = ~0x3f&(4*(BlockWidth+2*RADIUS)+0x3f);
                int sharedMem = SharedPitch*(threads.y+2*RADIUS);

                // for the shared kernel, width must be divisible by 4
                iw &= ~3;

                SobelShared<<<blocks, threads, sharedMem>>>((uchar4 *) odata,
                                                            iw,
#ifndef FIXED_BLOCKWIDTH
                                                            BlockWidth, SharedPitch,
#endif
                                                            iw, ih, fScale);
            }
            break;
    }

    checkCudaErrors(hipUnbindTexture(tex));
// rima - end approximable region
#pragma parrot.end("sobel")

}
